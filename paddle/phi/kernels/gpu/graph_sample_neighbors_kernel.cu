#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#ifdef PADDLE_WITH_HIP
#include <hip/hip_runtime.h>
#include <hiprand_kernel.h>
#else
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#endif

#include "paddle/phi/kernels/graph_sample_neighbors_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

constexpr int WARP_SIZE = 32;

template <typename T>
struct DegreeFunctor {
  const T* col_ptr;
  HOSTDEVICE explicit inline DegreeFunctor(const T* x) { this->col_ptr = x; }
  HOSTDEVICE inline int operator()(T i) const {
    return col_ptr[i + 1] - col_ptr[i];
  }
};

struct MaxFunctor {
  int cap;
  HOSTDEVICE explicit inline MaxFunctor(int cap) { this->cap = cap; }
  HOSTDEVICE inline int operator()(int x) const {
    if (x > cap) {
      return cap;
    }
    return x;
  }
};

template <typename T, int BLOCK_WARPS, int TILE_SIZE>
__global__ void SampleKernel(const uint64_t rand_seed,
                             int k,
                             const int64_t num_nodes,
                             const T* nodes,
                             const T* row,
                             const T* col_ptr,
                             T* output,
                             int* output_ptr,
                             int* output_idxs) {
  assert(blockDim.x == WARP_SIZE);
  assert(blockDim.y == BLOCK_WARPS);

  int64_t out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_nodes);
#ifdef PADDLE_WITH_HIP
  hiprandState rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x,
               threadIdx.y * WARP_SIZE + threadIdx.x,
               0,
               &rng);
#else
  hiprandState rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x,
              threadIdx.y * WARP_SIZE + threadIdx.x,
              0,
              &rng);
#endif

  while (out_row < last_row) {
    T node = nodes[out_row];
    T in_row_start = col_ptr[node];
    int deg = col_ptr[node + 1] - in_row_start;
    int out_row_start = output_ptr[out_row];

    if (deg <= k) {
      for (int idx = threadIdx.x; idx < deg; idx += WARP_SIZE) {
        output[out_row_start + idx] = row[in_row_start + idx];
      }
    } else {
      for (int idx = threadIdx.x; idx < k; idx += WARP_SIZE) {
        output_idxs[out_row_start + idx] = idx;
      }
#ifdef PADDLE_WITH_CUDA
      __syncwarp();
#endif

      for (int idx = k + threadIdx.x; idx < deg; idx += WARP_SIZE) {
#ifdef PADDLE_WITH_HIP
        const int num = hiprand(&rng) % (idx + 1);
#else
        const int num = hiprand(&rng) % (idx + 1);
#endif
        if (num < k) {
          atomicMax(reinterpret_cast<unsigned int*>(  // NOLINT
                        output_idxs + out_row_start + num),
                    static_cast<unsigned int>(idx));  // NOLINT
        }
      }
#ifdef PADDLE_WITH_CUDA
      __syncwarp();
#endif

      for (int idx = threadIdx.x; idx < k; idx += WARP_SIZE) {
        T perm_idx = output_idxs[out_row_start + idx] + in_row_start;
        output[out_row_start + idx] = row[perm_idx];
      }
    }

    out_row += BLOCK_WARPS;
  }
}

template <typename T, typename Context>
int GetTotalSampleNum(const thrust::device_ptr<const T> input,
                      const T* col_ptr,
                      thrust::device_ptr<int> output_count,
                      int sample_size,
                      int bs) {
  thrust::transform(input, input + bs, output_count, DegreeFunctor<T>(col_ptr));
  if (sample_size >= 0) {
    thrust::transform(
        output_count, output_count + bs, output_count, MaxFunctor(sample_size));
  }
  int total_sample_num = thrust::reduce(output_count, output_count + bs);
  return total_sample_num;
}

template <typename T, typename Context>
void SampleNeighbors(const Context& dev_ctx,
                     const T* row,
                     const T* col_ptr,
                     const thrust::device_ptr<const T> input,
                     thrust::device_ptr<T> output,
                     thrust::device_ptr<int> output_count,
                     int sample_size,
                     int bs,
                     int total_sample_num) {
  thrust::device_vector<int> output_ptr;
  thrust::device_vector<int> output_idxs;
  output_ptr.resize(bs);
  output_idxs.resize(total_sample_num);
  thrust::exclusive_scan(
      output_count, output_count + bs, output_ptr.begin(), 0);

  constexpr int BLOCK_WARPS = 128 / WARP_SIZE;
  constexpr int TILE_SIZE = BLOCK_WARPS * 16;
  const dim3 block(WARP_SIZE, BLOCK_WARPS);
  const dim3 grid((bs + TILE_SIZE - 1) / TILE_SIZE);
  SampleKernel<T, BLOCK_WARPS, TILE_SIZE><<<grid, block, 0, dev_ctx.stream()>>>(
      0,
      sample_size,
      bs,
      thrust::raw_pointer_cast(input),
      row,
      col_ptr,
      thrust::raw_pointer_cast(output),
      thrust::raw_pointer_cast(output_ptr.data()),
      thrust::raw_pointer_cast(output_idxs.data()));
}

template <typename T, typename Context>
void GraphSampleNeighborsKernel(const Context& dev_ctx,
                                const DenseTensor& row,
                                const DenseTensor& col_ptr,
                                const DenseTensor& x,
                                int sample_size,
                                DenseTensor* out,
                                DenseTensor* out_count) {
  auto* row_data = row.data<T>();
  auto* col_ptr_data = col_ptr.data<T>();
  auto* x_data = x.data<T>();
  int bs = x.dims()[0];

  const thrust::device_ptr<const T> input(x_data);

  out_count->Resize({bs});
  int* out_count_data = dev_ctx.template Alloc<int>(out_count);
  thrust::device_ptr<int> output_count(out_count_data);

  int total_sample_size = GetTotalSampleNum<T, Context>(
      input, col_ptr_data, output_count, sample_size, bs);

  out->Resize({static_cast<int>(total_sample_size)});
  T* out_data = dev_ctx.template Alloc<T>(out);
  thrust::device_ptr<T> output(out_data);

  SampleNeighbors<T, Context>(dev_ctx,
                              row_data,
                              col_ptr_data,
                              input,
                              output,
                              output_count,
                              sample_size,
                              bs,
                              total_sample_size);
}

}  // namespace phi

PD_REGISTER_KERNEL(graph_sample_neighbors,
                   GPU,
                   ALL_LAYOUT,
                   phi::GraphSampleNeighborsKernel,
                   int,
                   int64_t) {}
