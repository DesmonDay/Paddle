
#include <hip/hip_runtime.h>
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#if defined _WIN32 || defined __APPLE__
#else
#define _LINUX
#endif
#if defined(PADDLE_WITH_CUDA) && defined(PADDLE_WITH_HETERPS)

#include "paddle/fluid/framework/data_feed.h"
#include "paddle/fluid/framework/fleet/heter_ps/gpu_graph_node.h"
#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_wrapper.h"
#include "cub/cub.cuh"
namespace paddle {
namespace framework {

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

// CUDA: use 512 threads per block
const int CUDA_NUM_THREADS = 512;
// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}
// fill slot values
__global__ void FillSlotValueOffsetKernel(
    const int ins_num, const int used_slot_num, size_t *slot_value_offsets,
    const int *uint64_offsets, const int uint64_slot_size,
    const int *float_offsets, const int float_slot_size,
    const UsedSlotGpuType *used_slots) {
  int col_num = ins_num + 1;
  int uint64_cols = uint64_slot_size + 1;
  int float_cols = float_slot_size + 1;

  CUDA_KERNEL_LOOP(slot_idx, used_slot_num) {
    int value_off = slot_idx * col_num;
    slot_value_offsets[value_off] = 0;

    auto &info = used_slots[slot_idx];
    if (info.is_uint64_value) {
      for (int k = 0; k < ins_num; ++k) {
        int pos = k * uint64_cols + info.slot_value_idx;
        int num = uint64_offsets[pos + 1] - uint64_offsets[pos];
        PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
        slot_value_offsets[value_off + k + 1] =
            slot_value_offsets[value_off + k] + num;
      }
    } else {
      for (int k = 0; k < ins_num; ++k) {
        int pos = k * float_cols + info.slot_value_idx;
        int num = float_offsets[pos + 1] - float_offsets[pos];
        PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
        slot_value_offsets[value_off + k + 1] =
            slot_value_offsets[value_off + k] + num;
      }
    }
  }
}

void SlotRecordInMemoryDataFeed::FillSlotValueOffset(
    const int ins_num, const int used_slot_num, size_t *slot_value_offsets,
    const int *uint64_offsets, const int uint64_slot_size,
    const int *float_offsets, const int float_slot_size,
    const UsedSlotGpuType *used_slots) {
  auto stream =
      dynamic_cast<platform::CUDADeviceContext *>(
          paddle::platform::DeviceContextPool::Instance().Get(this->place_))
          ->stream();
  FillSlotValueOffsetKernel<<<GET_BLOCKS(used_slot_num), CUDA_NUM_THREADS, 0,
                              stream>>>(
      ins_num, used_slot_num, slot_value_offsets, uint64_offsets,
      uint64_slot_size, float_offsets, float_slot_size, used_slots);
  cudaStreamSynchronize(stream);
}

__global__ void CopyForTensorKernel(
    const int used_slot_num, const int ins_num, void **dest,
    const size_t *slot_value_offsets, const uint64_t *uint64_feas,
    const int *uint64_offsets, const int *uint64_ins_lens,
    const int uint64_slot_size, const float *float_feas,
    const int *float_offsets, const int *float_ins_lens,
    const int float_slot_size, const UsedSlotGpuType *used_slots) {
  int col_num = ins_num + 1;
  int uint64_cols = uint64_slot_size + 1;
  int float_cols = float_slot_size + 1;

  CUDA_KERNEL_LOOP(i, ins_num * used_slot_num) {
    int slot_idx = i / ins_num;
    int ins_idx = i % ins_num;

    uint32_t value_offset = slot_value_offsets[slot_idx * col_num + ins_idx];
    auto &info = used_slots[slot_idx];
    if (info.is_uint64_value) {
      uint64_t *up = reinterpret_cast<uint64_t *>(dest[slot_idx]);
      int index = info.slot_value_idx + uint64_cols * ins_idx;
      int old_off = uint64_offsets[index];
      int num = uint64_offsets[index + 1] - old_off;
      PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
      int uint64_value_offset = uint64_ins_lens[ins_idx];
      for (int k = 0; k < num; ++k) {
        up[k + value_offset] = uint64_feas[k + old_off + uint64_value_offset];
      }
    } else {
      float *fp = reinterpret_cast<float *>(dest[slot_idx]);
      int index = info.slot_value_idx + float_cols * ins_idx;
      int old_off = float_offsets[index];
      int num = float_offsets[index + 1] - old_off;
      PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
      int float_value_offset = float_ins_lens[ins_idx];
      for (int k = 0; k < num; ++k) {
        fp[k + value_offset] = float_feas[k + old_off + float_value_offset];
      }
    }
  }
}

void SlotRecordInMemoryDataFeed::CopyForTensor(
    const int ins_num, const int used_slot_num, void **dest,
    const size_t *slot_value_offsets, const uint64_t *uint64_feas,
    const int *uint64_offsets, const int *uint64_ins_lens,
    const int uint64_slot_size, const float *float_feas,
    const int *float_offsets, const int *float_ins_lens,
    const int float_slot_size, const UsedSlotGpuType *used_slots) {
  auto stream =
      dynamic_cast<platform::CUDADeviceContext *>(
          paddle::platform::DeviceContextPool::Instance().Get(this->place_))
          ->stream();

  CopyForTensorKernel<<<GET_BLOCKS(used_slot_num * ins_num), CUDA_NUM_THREADS,
                        0, stream>>>(
      used_slot_num, ins_num, dest, slot_value_offsets, uint64_feas,
      uint64_offsets, uint64_ins_lens, uint64_slot_size, float_feas,
      float_offsets, float_ins_lens, float_slot_size, used_slots);
  cudaStreamSynchronize(stream);
}

__global__ void GraphFillIdKernel(
    int64_t* id_tensor,
    int *actual_sample_size, int64_t *prefix_sum, int64_t* device_key,
    int64_t *neighbors, int sample_size, int len) {

  CUDA_KERNEL_LOOP(idx, len) {
    for (int k = 0; k < actual_sample_size[idx]; k++) {
      int offset = (prefix_sum[idx] + k) * 2;
      id_tensor[offset] = device_key[idx];
      id_tensor[offset + 1] = neighbors[idx * sample_size + k];
    }
  }
}

__global__ void GraphFillCVMKernel(int64_t* tensor, int len) {

  CUDA_KERNEL_LOOP(idx, len) {
    tensor[idx] = 1;
  }
}

void GraphDataGenerator::FeedGraphIns(size_t cursor, int len, NeighborSampleResult& sample_res) {
  size_t temp_storage_bytes = 0;
  int *d_actual_sample_size = sample_res.actual_sample_size;
  int64_t *d_neighbors = sample_res.val;
  int64_t* d_prefix_sum = reinterpret_cast<int64_t*>(d_prefix_sum_->ptr());
  CUDA_CHECK(cub::DeviceScan::ExclusiveSum(
              NULL, temp_storage_bytes, d_actual_sample_size, d_prefix_sum, len,
              stream_));
  auto d_temp_storage = memory::Alloc(place_, temp_storage_bytes);
  VLOG(0) << "temp_storage_bytes: " << temp_storage_bytes << " len: " << len;

  CUDA_CHECK(cub::DeviceScan::ExclusiveSum(
              d_temp_storage->ptr(), temp_storage_bytes, d_actual_sample_size, d_prefix_sum,
              len, stream_));
  cudaStreamSynchronize(stream_);
  int64_t total_ins = 0;
  cudaMemcpyAsync(&total_ins, d_prefix_sum + len, sizeof(int64_t), cudaMemcpyDeviceToHost, stream_);
  {
    int64_t *cpu_keys = new int64_t[len + 1];
    cudaMemcpy(cpu_keys, d_prefix_sum,
                 (len + 1) * sizeof(int64_t),
                 cudaMemcpyDeviceToHost);  // 3, 1, 3
  }
  total_ins *= 2;
  id_tensor_ptr_ = feed_vec_[0]->mutable_data<int64_t>({total_ins, 1}, this->place_);
  show_tensor_ptr_ = feed_vec_[1]->mutable_data<int64_t>({total_ins, 1}, this->place_);
  clk_tensor_ptr_ = feed_vec_[2]->mutable_data<int64_t>({total_ins, 1}, this->place_);

  GraphFillIdKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, stream_>>>(id_tensor_ptr_, d_actual_sample_size, d_prefix_sum, device_keys_ + cursor_, d_neighbors, walk_degree_, len);
  GraphFillCVMKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, stream_>>>(show_tensor_ptr_, total_ins);
  GraphFillCVMKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, stream_>>>(clk_tensor_ptr_, total_ins);
  
  offset_.clear();
  offset_.push_back(0);
  offset_.push_back(total_ins);
  LoD lod{offset_};
  feed_vec_[0]->set_lod(lod);
  feed_vec_[1]->set_lod(lod);
  feed_vec_[2]->set_lod(lod);
  cudaStreamSynchronize(stream_);
}

int GraphDataGenerator::GenerateBatch() {
  //GpuPsGraphTable *g = (GpuPsGraphTable *)(gpu_graph_ptr->graph_table);
  platform::CUDADeviceGuard guard(gpuid_);
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  int tmp_len = cursor_ + sample_key_size_ > device_key_size_ ? device_key_size_ - cursor_ : sample_key_size_;
  VLOG(3) << "device key size: " << device_key_size_ << " this batch: " << tmp_len << " cursor: " << cursor_ << " sample_key_size_: " << sample_key_size_;
  if (tmp_len == 0) {
    return 0;
  }
  int total_instance = 1;
  auto sample_res = gpu_graph_ptr->graph_neighbor_sample(gpuid_, device_keys_ + cursor_, walk_degree_, tmp_len);
  int64_t *cpu_keys = new int64_t[tmp_len];
  cudaMemcpy(cpu_keys, device_keys_ + cursor_,
             tmp_len * sizeof(int64_t),
             cudaMemcpyDeviceToHost);  // 3, 1, 3
  int *actual_sample_size = new int[tmp_len];
  cudaMemcpy(actual_sample_size, sample_res.actual_sample_size,
             tmp_len * sizeof(int),
             cudaMemcpyDeviceToHost);  // 3, 1, 3
  for (int i = 0; i < tmp_len; i++) {
   VLOG(0) << cpu_keys[i] << ":" << actual_sample_size[i];
  }
  free(actual_sample_size);
  free(cpu_keys);
  FeedGraphIns(cursor_, tmp_len, sample_res);
  cursor_ += tmp_len;
  return 1;
}

}  // namespace framework
}  // namespace paddle
#endif
